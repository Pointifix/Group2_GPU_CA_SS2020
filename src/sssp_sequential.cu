#include "hip/hip_runtime.h"
#include "sssp_sequential.h"

SSSP_Sequential::SSSP_Sequential(std::shared_ptr<Graph> graph) : SSSP(std::move(graph)) {
}

std::vector<std::vector<int>> SSSP_Sequential::compute(int source_node)
{
    int N = graph->edges.size();
    std::vector<int> cluster;
    std::vector<int> cost(N, std::numeric_limits<int>::max());
    std::vector<int> used_edge(N, -1);
    std::vector<int> prev_vert(N, -1);
    cost[source_node] = 0;
    prev_vert[source_node] = source_node;

    // we initially add the source node and update the costs
    cluster.push_back(source_node);

    // either we take all edges until the end or until the next node stores its edges
    size_t edge_list_end = (source_node + 1 < graph->edges.size()) ? graph->edges[source_node + 1] : graph->destinations.size();

    // for all edges incident from this node
    for(size_t i = graph->edges[source_node]; i < graph->destinations.size() && i < edge_list_end; i++)
    {
        // node that the edge is incident to (s) -> (dest)
        size_t dest_node = graph->destinations[i];
        cost[dest_node] = graph->weights[i]; // the cost is the weight of the edge
        used_edge[dest_node] = i; // we remember the index of the edge we use
        prev_vert[dest_node] = source_node; // we remember the node we came from for this edge
    }

    do
    {
        // pick the node with the smallest cost
        int min = std::numeric_limits<int>::max();
        int picked_node = -1;
        for(int i = 0; i < cost.size(); i++)
        {
            // if the cost is the smallest and we didn't pick this node yet
            if(cost[i] < min && std::find(cluster.begin(), cluster.end(), i) == cluster.end())
            {
                picked_node = i;
                min = cost[i];
            }
        }

        // we cannot pick node -1 lets stop or burn and crash
        if(picked_node == -1)
        {
            break;
        }

        // we add this node to the cluster (index of the picked node)
        cluster.push_back(picked_node);
        // we select the edge-list (index of the edge list)
        size_t edge_list_index_start = graph->edges[picked_node];

        // either we take all edges until the end or until the next node stores its edges
        // source_node+1 = following node if it exists
        edge_list_end = (picked_node + 1 < graph->edges.size()) ? graph->edges[picked_node + 1] : graph->destinations.size();

        // we update the costs for all edges from this node
        // until we reach the end of the destinations array or the edges of the next node start
        for(size_t i = edge_list_index_start; i < graph->destinations.size() && i < edge_list_end; i++)
        {
            // the node targeted by this edge
            size_t dest_node = graph->destinations[i];

            // if we already picked it we don't need to do anything, continue with the next one
            if(std::find(cluster.begin(), cluster.end(), dest_node) != cluster.end()) {
                continue;
            }

            // cost to the picked node + weight of the current edge < cost for the destined node ?
            if(cost[picked_node] + graph->weights[i] < cost[dest_node])
            {
                cost[dest_node] = cost[picked_node] + graph->weights[i]; // lets update its cost
                used_edge[dest_node] = i;


                prev_vert[dest_node] = picked_node;// we remember the node we came from for this edge
            }
        }
    } while(cluster.size() < graph->edges.size()); // while there is a vertex that is not reached yet

    std::vector<std::vector<int>> shortest_paths;

    // to find all shortest paths we can just traverse the cluster reversed
    for(int i = cluster.size()-1; i > 0; i--)
    {
        std::vector<int> path;
        int node = cluster[i];
        int prev = prev_vert[node];

        path.push_back(node);

        // as long as the previous node is not the source node we keep iterating
        while(prev != source_node)
        {
            path.push_back(prev);
            node = prev;
            prev = prev_vert[node];
        }

        // for completeness we also add the source node
        path.push_back(prev);
        shortest_paths.push_back(path);
    }

    // as the paths are now reversed, we simply reverse the vectors
    std::reverse(shortest_paths.begin(), shortest_paths.end());
    for(int i = 0; i < shortest_paths.size(); i++)
    {
        std::reverse(shortest_paths[i].begin(), shortest_paths[i].end());
    }

    std::cout << "Debug output:" << std::endl;
    for(int i = 0; i < shortest_paths.size(); i++)
    {
        std::cout << "Paths [" << source_node << "] to [" << shortest_paths[i][shortest_paths[i].size()-1] << "]: ";
        for(int j = 0; j < shortest_paths[i].size(); j++)
        {
            std::cout << " " << shortest_paths[i][j] << ",";
        }
        std::cout << std::endl;
    }

    return shortest_paths;
}