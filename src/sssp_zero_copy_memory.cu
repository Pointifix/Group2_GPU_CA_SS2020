#include "hip/hip_runtime.h"
#include "sssp_zero_copy_memory.h"

#include <utility>

SSSP_Zero_Copy_Memory::SSSP_Zero_Copy_Memory(std::shared_ptr<Graph> graph) : SSSP(std::move(graph)) {
}

std::shared_ptr<Paths> SSSP_Zero_Copy_Memory::compute(int source_node)
{
    size_t numNodes = graph->edges.size();
    size_t numEdges = graph->destinations.size();

    // Sizes
    size_t sizeNodes = numNodes * sizeof(pos_t);
    size_t sizeEdges = numEdges * sizeof(pos_t);
    size_t sizeWeights = numEdges * sizeof(weight_t);
    size_t sizeMask = numNodes * sizeof(mask_t);
    size_t sizeCost = numNodes * sizeof(weight_t);

    // Device memory
    pos_t *d_edges = nullptr;
    pos_t *d_destinations = nullptr;
    weight_t *d_weights = nullptr;
    mask_t *d_mask = nullptr;
    pos_t *d_previous_node = nullptr;
    weight_t *d_cost = nullptr;

    mask_t *mask = nullptr;
    pos_t *previous_nodes = nullptr;
    weight_t *cost = nullptr;

    // Allocate host memory for mask
    M_C(hipHostAlloc(&mask, sizeMask, hipHostMallocMapped));
    M_C(hipHostGetDevicePointer(&d_mask, mask, 0)); // Map mask to d_mask
    previous_nodes = new pos_t[numNodes];
    cost = new weight_t[numNodes];

    // No need to allocate d_mask in Zero Copy mode!
    M_C(hipMalloc((void **) &d_previous_node, sizeNodes));
    M_C(hipMalloc((void **) &d_cost, sizeCost));

    M_C(hipMalloc((void **) &d_edges, sizeNodes));
    M_C(hipMalloc((void **) &d_destinations, sizeEdges));
    M_C(hipMalloc((void **) &d_weights, sizeWeights));
    M_C(hipMemcpy(d_edges, graph->edges.data(), sizeNodes, hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_destinations, graph->destinations.data(), sizeEdges, hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_weights, graph->weights.data(), sizeWeights, hipMemcpyHostToDevice));

    alg::fill_parcu(d_mask, numNodes, M_MASK_FALSE);
    alg::fill_parcu(d_previous_node, numNodes, M_INVALID_POSITION);
    alg::fill_parcu(d_cost, numNodes, std::numeric_limits<weight_t>::max());

    alg::set_parcu(d_mask, source_node, M_MASK_TRUE);
    alg::set_parcu(d_cost, source_node, 0);

    // while we still find true in the mask (Ma not empty)
    const mask_t *maskFirst = &mask[0];
    const mask_t *maskLast = &mask[numNodes];
    do {
        int numBlocks = ceil((double) graph->edges.size() / M_BLOCKSIZE);
        M_CFUN((alg::SSSP_Kernel<<<numBlocks, M_BLOCKSIZE>>>(d_edges, d_destinations, d_weights,
                       d_previous_node, d_mask, d_cost, graph->edges.size(), graph->destinations.size())));
        M_C(hipDeviceSynchronize());
    } while (std::find(maskFirst, maskLast, true) != maskLast);

    M_C(hipMemcpy(previous_nodes, d_previous_node, sizeNodes, hipMemcpyDeviceToHost));
    M_C(hipMemcpy(cost, d_cost, sizeCost, hipMemcpyDeviceToHost));
    std::vector<pos_t> ret_previous_nodes(previous_nodes, previous_nodes + graph->edges.size());
    std::vector<weight_t> ret_cost(cost, cost + graph->edges.size());

    M_C(hipFree(d_edges));
    M_C(hipFree(d_destinations));
    M_C(hipFree(d_weights));
    M_C(hipFree(d_previous_node));
    M_C(hipFree(d_cost));

    M_C(hipHostFree(mask));
    delete previous_nodes;
    delete cost;

    std::shared_ptr<Paths> paths = std::make_shared<Paths>(Paths(ret_previous_nodes, ret_cost, source_node, graph));

    return paths;
}