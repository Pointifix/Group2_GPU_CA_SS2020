#include "hip/hip_runtime.h"
#include "graph.h"

 /**
  * Generates a graph instance.
  * @param edges Edges
  * @param destinations Destinations
  * @param weights Weights
  */
Graph::Graph(std::vector<pos_t> &edges, std::vector<pos_t> &destinations, std::vector<weight_t> &weights) :
    edges(std::move(edges)), destinations(std::move(destinations)), weights(std::move(weights))
{
}

std::string Graph::toString() {
    std::string builder = "Graph (" + std::to_string(edges.size()) + " vertices, " + std::to_string(destinations.size()) + " edges)";

    builder += "\nEdges:\n\t";
    for (const auto &edge : edges) {
        builder += std::to_string(edge) + ",\t";
    }
    builder += "\nDestinations:\n\t";
    for (const auto &destination : destinations) {
        builder += std::to_string(destination) + ",\t";
    }
    builder += "\nWeights:\n\t";
    for (const auto &weight : weights) {
        builder += std::to_string(weight) + ",\t";
    }
    return builder + "\n";
}

std::vector<std::vector<weight_t>> Graph::printAdjacencyMatrix()
{
    std::vector<std::vector<weight_t>> adjacencyMatrix(edges.size(), std::vector<weight_t>(edges.size(), 0));

    for(pos_t i = 0; i < edges.size(); i++)
    {
        pos_t first = edges[i];
        pos_t last = (i + 1 < edges.size()) ? edges[i + 1] : destinations.size();

        for(pos_t j = first; j < last; j++)
        {
            adjacencyMatrix[i][destinations[j]] = weights[j];
        }
    }

    for(pos_t i = 0; i < edges.size(); i++)
    {
        for(pos_t j = 0; j < edges.size(); j++)
        {
            std::cout << adjacencyMatrix[i][j] << ", ";
        }
        std::cout << std::endl;
    }

    return adjacencyMatrix;
}

Paths::Paths(std::vector<pos_t> &previous_nodes, std::vector<weight_t> &costs, pos_t source_node, std::shared_ptr<Graph> graph) :
    previous_nodes(std::move(previous_nodes)), costs(costs), source_node(source_node), graph(graph)
{
}

std::string Paths::toString() {
    std::string builder = "Paths (Graph with " + std::to_string(graph->edges.size()) + " vertices, " + std::to_string(graph->destinations.size()) + " edges)";

    builder += "\nSource Node:\n\t" + std::to_string(source_node);

    builder += "\nPrevious Nodes:\n\t";
    for (const auto &previous_node : previous_nodes) {
        builder += std::to_string(previous_node) + ",\t";
    }
    builder += "\nCosts:\n\t";
    for (const auto &cost : costs) {
        if (cost == std::numeric_limits<int>::max()) builder += "inf,\t";
        else builder += std::to_string(cost) + ",\t";
    }
    return builder + "\n";
}

std::vector<pos_t> Paths::getPath(pos_t destination)
{
    std::vector<pos_t> path;
    path.push_back(destination);

    pos_t current_node = destination;

    while(previous_nodes[current_node] != -1 && current_node != source_node)
    {
        current_node = previous_nodes[current_node];
        path.push_back(current_node);
    }

    std::reverse(std::begin(path), std::end(path));

    return path;
}

int Paths::isEqualTo(const Paths* path) {
    if (this->previous_nodes.size() != path->previous_nodes.size()) return -2;

    std::vector<int> difference;

    for (size_t i = 0; i < this->previous_nodes.size(); i++)
    {
        if (this->previous_nodes.at(i) != path->previous_nodes.at(i)) difference.push_back(i);
    }

    for (pos_t i = 0; i < difference.size(); i++)
    {
        if (this->costs.at(difference.at(i)) != path->costs.at(difference.at(i))) return -1;
    }

    return difference.size();
}
