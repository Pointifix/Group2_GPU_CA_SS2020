#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "alg.cuh"
#include "graph.h"
#include "graph_generator.h"
#include "graph_io.h"
#include "sssp.h"
#include "sssp_sequential.h"
#include "sssp_standard.h"
#include "sssp_thrust.h"
#include "time_measurement.h"
#include "sssp_pinned_memory.h"
#include "sssp_zero_copy_memory.h"
#include "sssp_gpu_search.h"

#ifdef  DEBUG
void test();
#endif

int main()
{
    // Enable Zero Copy
    // Source: https://arrayfire.com/zero-copy-on-tegra-k1/
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);
    if (!prop.canMapHostMemory) {
        M_RUNTIME_ERROR("Zero copy memory not supported by the GPU");
    }
    hipSetDeviceFlags(hipDeviceMapHost);

#ifdef DEBUG
    test();
    //return 0;
#endif

    srand(time(nullptr));

    for (int i = 1; i <= 8; i++)
    {
        int nodes = pow(10, i);

        time_measurement::startMeasurement("Graph Generation");
        std::shared_ptr<Graph> graph = graphgen::generateGraph(nodes, graphgen::calculateDensity(nodes * 5, nodes, true));
        time_measurement::endMeasurement("Graph Generation");

        //std::cout << graph->toString() << std::endl;

        /*
        time_measurement::startMeasurement("Graph Output");
        graphio::writeGraph("output/graph", graph);
        time_measurement::endMeasurement("Graph Output");

        time_measurement::startMeasurement("Graph Input");
        std::shared_ptr<Graph> graph2 = graphio::readGraph("output/graph");
        time_measurement::endMeasurement("Graph Input");
         */


        int random_source = rand() % nodes;

        SSSP_Sequential sequ(graph);
        time_measurement::startMeasurement("SSSP Sequential");
        std::shared_ptr<Paths> paths1 = sequ.compute(random_source);
        time_measurement::endMeasurement("SSSP Sequential");

        //graphio::writePaths("output/path_sequential", paths1);

        SSSP_Standard standard(graph);
        time_measurement::startMeasurement("SSSP Standard");
        std::shared_ptr<Paths> paths2 = standard.compute(random_source);
        time_measurement::endMeasurement("SSSP Standard");

        //graphio::writePaths("output/path_standard", paths2);

        SSSP_Thrust thrust(graph);
        time_measurement::startMeasurement("SSSP Thrust");
        std::shared_ptr<Paths> paths3 = thrust.compute(random_source);
        time_measurement::endMeasurement("SSSP Thrust");

        SSSP_Pinned_Memory pinned(graph);
        time_measurement::startMeasurement("SSSP Pinned");
        std::shared_ptr<Paths> paths4 = pinned.compute(random_source);
        time_measurement::endMeasurement("SSSP Pinned");

        SSSP_Zero_Copy_Memory zeroCopy(graph);
        time_measurement::startMeasurement("SSSP Zero Copy");
        std::shared_ptr<Paths> paths5 = zeroCopy.compute(random_source);
        time_measurement::endMeasurement("SSSP Zero Copy");

        SSSP_GPU_Search gpuSearch(graph);
        time_measurement::startMeasurement("SSSP GPU Search");
        std::shared_ptr<Paths> paths6 = gpuSearch.compute(random_source);
        time_measurement::endMeasurement("SSSP GPU Search");

        std::cout << "path 1 and 2 same? " << paths1->isEqualTo(paths2.get()) << std::endl;
        std::cout << "path 2 and 3 same? " << paths2->isEqualTo(paths3.get()) << std::endl;
        std::cout << "path 1 and 3 same? " << paths1->isEqualTo(paths3.get()) << std::endl;
        std::cout << "path 1 and 4 same? " << paths1->isEqualTo(paths4.get()) << std::endl;
        std::cout << "path 1 and 5 same? " << paths1->isEqualTo(paths5.get()) << std::endl;
        std::cout << "path 1 and 6 same? " << paths1->isEqualTo(paths6.get()) << std::endl;

        std::cout << "\nGraph (" << graph->edges.size() << " Vertices, "<< graph->destinations.size() << " Edges)" << std::endl;
    }
    time_measurement::printMeasurements();
}

#ifdef DEBUG
void test() {
    M_A(true); // Use M_A to make assertions!
}
#endif