#include "hip/hip_runtime.h"
#include "alg.cuh"
#include "common.cuh"

namespace alg {

    void countoccur_seq(const std::vector<uint> &a, std::vector<uint> &out) {
        for (uint v : a) {
            out[v]++;
        }
    }

    void exscan_seq(const std::vector<uint> &a, std::vector<uint> &out) {
        M_A(a.size() == out.size());
        if (a.empty()) return;

        out[0] = 0;
        for (int i = 1; i < a.size(); i++) {
            out[i] = out[i-1] + a[i-1];
        }
    }

    // -----------------------------------------------------------------------------------------------------------------
    // PARALLEL IMPLEMENTATIONS (CUDA)
    // -----------------------------------------------------------------------------------------------------------------

    __global__ void _countoccur_parcu(const uint *a, int Na, uint *out) {
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i < Na) {
            uint val = a[i];
            atomicAdd(&out[val], 1u); // Increment
        }
    }
    void countoccur_parcu(const std::vector<uint> &a, std::vector<uint> &out) {
        if (a.empty()) return;
        if (out.empty()) return;

        uint *d_a, *d_out;
        size_t sizea = a.size() * sizeof(uint);
        size_t sizeout = out.size() * sizeof(uint);

        int threadsPerBlock = M_BLOCKSIZE;
        int numBlocks = (int) ceil((float)a.size() / (float)threadsPerBlock);

        M_C(hipMalloc((void **) &d_a, sizea));
        M_C(hipMalloc((void **) &d_out, sizeout));
        {
            M_C(hipMemcpy(d_a, a.data(), sizea, hipMemcpyHostToDevice));
            M_CFUN((_countoccur_parcu<<< numBlocks, threadsPerBlock >>>(d_a, a.size(), d_out)));
            M_C(hipMemcpy(out.data(), d_out, sizeout, hipMemcpyDeviceToHost));
        }
        M_C(hipFree(d_a));
        M_C(hipFree(d_out));
    }

    __global__ void _add_parcu(const uint *a, const uint *b, uint *out, int N) {
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i < N) {
            out[i] = a[i] + b[i];
        }
    }
    void add_parcu(const std::vector<uint> &a,
                   const std::vector<uint> &b,
                   std::vector<uint> &out) {
        M_A(a.size() == b.size());
        M_A(a.size() == out.size());
        if (a.empty()) return;

        size_t N = a.size();
        const size_t size = N * sizeof(uint);

        int threadsPerBlock = M_BLOCKSIZE;
        int numBlocks = (int) ceil((float)N / (float)threadsPerBlock);

        uint *d_a, *d_b, *d_out;

        M_C(hipMalloc((void **) &d_a, size));
        M_C(hipMalloc((void **) &d_b, size));
        M_C(hipMalloc((void **) &d_out, size));
        {
            M_C(hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice));
            M_C(hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice));
            M_CFUN((_add_parcu<<< numBlocks, threadsPerBlock >>>(d_a, d_b, d_out, N)));
            M_C(hipMemcpy(out.data(), d_out, size, hipMemcpyDeviceToHost));
        }
        M_C(hipFree(d_a));
        M_C(hipFree(d_b));
        M_C(hipFree(d_out));
    }

    /**
     * Source: https://www.eecs.umich.edu/courses/eecs570/hw/parprefix.pdf
     * @param a
     * @param out
     * @param N
     */
    __global__ void _exscan_parcu(const uint *a, uint *out, int N) {
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i < N) {

            // Up-sweep (reduce)
            

        }
    }
    void exscan_parcu(const std::vector<uint> &a, std::vector<uint> &out) {
        M_A(a.size() == out.size());
        if (a.empty()) return;

        size_t N = a.size();
        const size_t size = N * sizeof(uint);

        int threadsPerBlock = M_BLOCKSIZE;
        int numBlocks = (int) ceil((float)N / (float)threadsPerBlock);


    }

}