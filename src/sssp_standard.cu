#include "hip/hip_runtime.h"
#include "sssp_standard.h"

SSSP_Standard::SSSP_Standard(std::shared_ptr<Graph> graph) : SSSP(std::move(graph)) {
}



std::shared_ptr<Paths> SSSP_Standard::compute(int source_node)
{
    std::vector<m_t> previous_nodes(graph->edges.size(), -1);
    std::vector<int> mask(graph->edges.size(), 0);
    std::vector<m_t> cost(graph->edges.size(), std::numeric_limits<m_t>::max());

    mask.at(source_node) = true;
    cost.at(source_node) = 0;

    m_t *d_edges = nullptr;
    m_t *d_destinations = nullptr;
    m_t *d_weights = nullptr;
    m_t *d_previous_node = nullptr;
    int *d_mask = nullptr;
    m_t *d_cost = nullptr;

    M_C(hipMalloc((void**) &d_edges,          graph->edges.size() * sizeof(m_t)));
    M_C(hipMalloc((void**) &d_destinations,   graph->destinations.size() * sizeof(m_t)));
    M_C(hipMalloc((void**) &d_weights,        graph->weights.size() * sizeof(m_t)));

    M_C(hipMalloc((void**) &d_previous_node, previous_nodes.size() * sizeof(m_t)));
    M_C(hipMalloc((void**) &d_mask, mask.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_cost, cost.size() * sizeof(m_t)));

    M_C(hipMemcpy(d_edges,        &graph->edges[0],        graph->edges.size() * sizeof(m_t),          hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_destinations, &graph->destinations[0], graph->destinations.size() * sizeof(m_t),   hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_weights,      &graph->weights[0],      graph->weights.size() * sizeof(m_t),        hipMemcpyHostToDevice));

    M_C(hipMemcpy(d_previous_node, previous_nodes.data(),  previous_nodes.size() * sizeof(m_t),hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_mask,          mask.data(),            mask.size() * sizeof(int),         hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_cost,          cost.data(),            cost.size() * sizeof(m_t),          hipMemcpyHostToDevice));

    // while we still find false in the mask (Ma not empty)
    while (std::find(mask.begin(), mask.end(), true) != mask.end())
    {
        int numBlocks = ceil((double)graph->edges.size() / M_BLOCKSIZE);

        M_CFUN((alg::SSSP_Kernel<<<numBlocks, M_BLOCKSIZE>>>(d_edges, d_destinations, d_weights,
                d_previous_node, d_mask, d_cost, graph->edges.size(), graph->destinations.size())));

        //copy back mask
        M_C(hipMemcpy(&mask[0], d_mask, mask.size() * sizeof(int), hipMemcpyDeviceToHost));
    }

    M_C(hipMemcpy(&previous_nodes[0], d_previous_node, previous_nodes.size() * sizeof(int), hipMemcpyDeviceToHost));
    M_C(hipMemcpy(&cost[0], d_cost, cost.size() * sizeof(int), hipMemcpyDeviceToHost));

    M_C(hipFree(d_edges));
    M_C(hipFree(d_destinations));
    M_C(hipFree(d_weights));
    M_C(hipFree(d_previous_node));
    M_C(hipFree(d_mask));
    M_C(hipFree(d_cost));

    std::shared_ptr<Paths> paths = std::make_shared<Paths>(Paths(previous_nodes, cost, source_node, graph));

    return paths;
}