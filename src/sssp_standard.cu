#include "hip/hip_runtime.h"
#include "sssp_standard.h"

SSSP_Standard::SSSP_Standard(std::shared_ptr<Graph> graph) : SSSP(std::move(graph)) {
}

std::shared_ptr<Paths> SSSP_Standard::compute(int source_node) {
    size_t numNodes = graph->edges.size();
    size_t numEdges = graph->destinations.size();

    // Sizes
    size_t sizeNodes = numNodes * sizeof(pos_t);
    size_t sizeEdges = numEdges * sizeof(pos_t);
    size_t sizeWeights = numEdges * sizeof(weight_t);
    size_t sizeMask = numNodes * sizeof(mask_t);
    size_t sizeCost = numNodes * sizeof(weight_t);

    // Device memory
    pos_t *d_edges = nullptr;
    pos_t *d_destinations = nullptr;
    weight_t *d_weights = nullptr;
    mask_t *d_mask = nullptr;
    pos_t *d_previous_node = nullptr;
    weight_t *d_cost = nullptr;

    // Allocate host memory
    auto *mask = new mask_t[numNodes];
    auto *previous_nodes = new pos_t[numNodes];
    auto *cost = new weight_t[numNodes];

    // Allocate d_previous_node and d_cost no matter the mode
    M_C(hipMalloc((void **) &d_previous_node, sizeNodes));
    M_C(hipMalloc((void **) &d_cost, sizeCost));
    M_C(hipMalloc((void **) &d_mask, sizeMask));

    M_C(hipMalloc((void **) &d_edges, sizeNodes));
    M_C(hipMalloc((void **) &d_destinations, sizeEdges));
    M_C(hipMalloc((void **) &d_weights, sizeWeights));
    M_C(hipMemcpy(d_edges, graph->edges.data(), sizeNodes, hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_destinations, graph->destinations.data(), sizeEdges, hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_weights, graph->weights.data(), sizeWeights, hipMemcpyHostToDevice));

    alg::fill_parcu(d_mask, numNodes, M_MASK_FALSE);
    alg::fill_parcu(d_previous_node, numNodes, M_INVALID_POSITION);
    alg::fill_parcu(d_cost, numNodes, std::numeric_limits<weight_t>::max());

    alg::set_parcu(d_mask, source_node, M_MASK_TRUE);
    alg::set_parcu(d_cost, source_node, 0);

    // while we still find true in the mask (Ma not empty)
    const mask_t *maskFirst = &mask[0];
    const mask_t *maskLast = &mask[numNodes];
    do {
        int numBlocks = ceil((double) graph->edges.size() / M_BLOCKSIZE);

        M_CFUN((alg::SSSP_Kernel<<<numBlocks, M_BLOCKSIZE>>>(d_edges, d_destinations, d_weights,
                       d_previous_node, d_mask, d_cost, graph->edges.size(), graph->destinations.size())));

        //copy back mask
        M_C(hipMemcpy(mask, d_mask, sizeMask, hipMemcpyDeviceToHost));
    } while (std::find(maskFirst, maskLast, true) != maskLast);

    M_C(hipMemcpy(previous_nodes, d_previous_node, sizeNodes, hipMemcpyDeviceToHost));
    M_C(hipMemcpy(cost, d_cost, sizeCost, hipMemcpyDeviceToHost));
    std::vector<pos_t> ret_previous_nodes(previous_nodes, previous_nodes + graph->edges.size());
    std::vector<weight_t> ret_cost(cost, cost + graph->edges.size());

    M_C(hipFree(d_edges));
    M_C(hipFree(d_destinations));
    M_C(hipFree(d_weights));
    M_C(hipFree(d_previous_node));
    M_C(hipFree(d_cost));
    M_C(hipFree(d_mask));

    delete[] mask;
    delete[] previous_nodes;
    delete[] cost;

    std::shared_ptr<Paths> paths = std::make_shared<Paths>(Paths(ret_previous_nodes, ret_cost, source_node, graph));

    return paths;
}