#include "hip/hip_runtime.h"
#include "sssp_standard.h"

SSSP_Standard::SSSP_Standard(std::shared_ptr<Graph> graph) : SSSP(std::move(graph)) {
}



std::shared_ptr<Paths> SSSP_Standard::compute(int source_node)
{
    std::vector<int> previous_nodes(graph->edges.size(), -1);
    std::vector<int> mask(graph->edges.size(), 0);
    std::vector<int> cost(graph->edges.size(), std::numeric_limits<int>::max());

    mask.at(source_node) = true;
    cost.at(source_node) = 0;

    int *d_edges = nullptr;
    int *d_destinations = nullptr;
    int *d_weights = nullptr;
    int *d_previous_node = nullptr;
    int *d_mask = nullptr;
    int *d_cost = nullptr;

    M_C(hipMalloc((void**) &d_edges,          graph->edges.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_destinations,   graph->destinations.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_weights,        graph->weights.size() * sizeof(int)));

    M_C(hipMalloc((void**) &d_previous_node, previous_nodes.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_mask, mask.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_cost, cost.size() * sizeof(int)));

    M_C(hipMemcpy(d_edges,        &graph->edges[0],        graph->edges.size() * sizeof(int),          hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_destinations, &graph->destinations[0], graph->destinations.size() * sizeof(int),   hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_weights,      &graph->weights[0],      graph->weights.size() * sizeof(int),        hipMemcpyHostToDevice));

    M_C(hipMemcpy(d_previous_node,&previous_nodes[0],  previous_nodes.size() * sizeof(int),hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_mask,         &mask[0],            mask.size() * sizeof(int),          hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_cost,         &cost[0],            cost.size() * sizeof(int),          hipMemcpyHostToDevice));

    // while we still find false in the mask (Ma not empty)
    while (std::find(mask.begin(), mask.end(), true) != mask.end())
    {
        int numBlocks = ceil((double)graph->edges.size() / 256);

        dim3 threadsPerBlock(256);
        M_CFUN((alg::SSSP_Kernel<<<numBlocks, threadsPerBlock>>>(d_edges, d_destinations, d_weights,
                d_previous_node, d_mask, d_cost, graph->edges.size(), graph->destinations.size())));

        //copy back mask
        M_C(hipMemcpy(&mask[0], d_mask, mask.size() * sizeof(int), hipMemcpyDeviceToHost));
    }

    M_C(hipMemcpy(&previous_nodes[0], d_previous_node, previous_nodes.size() * sizeof(int), hipMemcpyDeviceToHost));
    M_C(hipMemcpy(&cost[0], d_cost, cost.size() * sizeof(int), hipMemcpyDeviceToHost));

    M_C(hipFree(d_edges));
    M_C(hipFree(d_destinations));
    M_C(hipFree(d_weights));
    M_C(hipFree(d_previous_node));
    M_C(hipFree(d_mask));
    M_C(hipFree(d_cost));

    std::shared_ptr<Paths> paths = std::make_shared<Paths>(Paths(previous_nodes, cost, source_node, graph));

    return paths;
}