#include "hip/hip_runtime.h"
#include "sssp_standard.h"

SSSP_Standard::SSSP_Standard(std::shared_ptr<Graph> graph, MemoryType memType, SearchType searchType) :
    SSSP(std::move(graph)), m_memType(memType), m_searchType(searchType) {
}

std::shared_ptr<Paths> SSSP_Standard::compute(int source_node)
{
    size_t numNodes = graph->edges.size();
    size_t numEdges = graph->destinations.size();

    size_t sizeNodes = numNodes * sizeof(pos_t);
    size_t sizeEdges = numEdges * sizeof(pos_t);
    size_t sizeWeights = numEdges * sizeof(weight_t);
    size_t sizeMask = numNodes * sizeof(mask_t);
    size_t sizeCost = numNodes * sizeof(weight_t);

    // Device memory
    pos_t *d_edges = nullptr;
    pos_t *d_destinations = nullptr;
    weight_t *d_weights = nullptr;
    mask_t *d_mask = nullptr;
    pos_t *d_previous_node = nullptr;
    weight_t *d_cost = nullptr;

    mask_t *mask = nullptr;
    std::function<bool()> maskContainsTrue;
    switch(m_searchType) {
        case GPU:
            // TODO
            M_A(false);
            std::terminate();
            break;
        case CPU:
        default:
            if (m_memType == ZERO_COPY) {
                M_C(hipHostAlloc(&mask, sizeMask, hipHostMallocMapped));
            } else if (m_memType == PINNED) {
                M_C(hipHostMalloc((void **) &mask, sizeMask));
            } else { // including memType NORMAL
                mask = new bool[numNodes];
            }

            const mask_t *maskFirst = &mask[0];
            const mask_t *maskLast = &mask[numNodes];
            maskContainsTrue = [maskFirst, maskLast]() {
                return std::find(maskFirst, maskLast, true) != maskLast;
            };
            break;
    }

    if (m_memType == ZERO_COPY) {
        hipHostGetDevicePointer(&d_mask, mask, 0);
    } else {
        M_C(hipMalloc((void**) &d_mask,          sizeMask));
    }

    M_C(hipMalloc((void**) &d_edges,         sizeNodes));
    M_C(hipMalloc((void**) &d_destinations,  sizeEdges));
    M_C(hipMalloc((void**) &d_weights,       sizeWeights));
    M_C(hipMalloc((void**) &d_previous_node, sizeNodes));
    M_C(hipMalloc((void**) &d_cost,          sizeCost));

    M_C(hipMemcpy(d_edges, graph->edges.data(), sizeNodes, hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_destinations, graph->destinations.data(), sizeEdges, hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_weights, graph->weights.data(), sizeWeights, hipMemcpyHostToDevice));

    alg::fill_parcu(d_mask, numNodes, M_MASK_FALSE);
    alg::fill_parcu(d_previous_node, numNodes, M_INVALID_POSITION);
    alg::fill_parcu(d_cost, numNodes, std::numeric_limits<weight_t>::max());

    alg::set_parcu(d_mask, source_node, M_MASK_TRUE);
    alg::set_parcu(d_cost, source_node, 0);

    // while we still find true in the mask (Ma not empty)
    do
    {
        int numBlocks = ceil((double)graph->edges.size() / M_BLOCKSIZE);

        M_CFUN((alg::SSSP_Kernel<<<numBlocks, M_BLOCKSIZE>>>(d_edges, d_destinations, d_weights,
                d_previous_node, d_mask, d_cost, graph->edges.size(), graph->destinations.size())));

        if (m_searchType == CPU && m_memType != ZERO_COPY) {
            //copy back mask
            M_C(hipMemcpy(mask, d_mask, sizeMask, hipMemcpyDeviceToHost));
        }
    }
    while (maskContainsTrue());

    std::vector<pos_t> previous_nodes(numNodes);
    std::vector<weight_t> cost(numNodes);
    M_C(hipMemcpy(previous_nodes.data(), d_previous_node, sizeNodes, hipMemcpyDeviceToHost));
    M_C(hipMemcpy(cost.data(), d_cost, sizeCost, hipMemcpyDeviceToHost));

    M_C(hipFree(d_edges));
    M_C(hipFree(d_destinations));
    M_C(hipFree(d_weights));
    M_C(hipFree(d_previous_node));
    M_C(hipFree(d_cost));

    if (m_memType != ZERO_COPY) {
        M_C(hipFree(d_mask));
    }

    if (m_searchType == CPU && m_memType == NORMAL) {
        delete mask;
    }

    std::shared_ptr<Paths> paths = std::make_shared<Paths>(Paths( previous_nodes, cost, source_node, graph));

    return paths;
}