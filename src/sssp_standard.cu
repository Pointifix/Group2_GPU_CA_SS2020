#include "hip/hip_runtime.h"
#include "sssp_standard.h"

SSSP_Standard::SSSP_Standard(std::shared_ptr<Graph> graph) : SSSP(std::move(graph)) {
}

//https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
__device__ int getGlobalIdx_3D_3D(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                  + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                   + (threadIdx.z * (blockDim.x * blockDim.y))
                   + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void CUDA_SSSP_Kernel1(const int* edges, const int* destinations, const int* weights, int* previous_node, int* mask,
        const int* cost, int* update_cost, int nodes_amount, int edges_amount)
{
    int tid = getGlobalIdx_3D_3D();

    if (tid >= nodes_amount) return;

    if (mask[tid])
    {
        int first = edges[tid];
        int last = (tid + 1 < nodes_amount) ? edges[tid + 1] : edges_amount;

        mask[tid] = false;

        for (int i = first; i < last; i++)
        {
            int nid = destinations[i];

            if(update_cost[nid] > cost[tid] + weights[i])
            {
                update_cost[nid] = cost[tid] + weights[i];
                previous_node[nid] = tid;
            }
        }
    }
}

__global__ void CUDA_SSSP_Kernel2(int* mask, int* cost, int* update_cost, int nodes_amount)
{
    int tid = getGlobalIdx_3D_3D();

    if (tid >= nodes_amount) return;

    if(cost[tid] > update_cost[tid])
    {
        cost[tid] = update_cost[tid];
        mask[tid] = true;
    }

    update_cost[tid] = cost[tid];
}

std::shared_ptr<Paths> SSSP_Standard::compute(int source_node)
{
    std::vector<int> previous_nodes(graph->edges.size(), -1);
    std::vector<int> mask(graph->edges.size(), 0);
    std::vector<int> cost(graph->edges.size(), std::numeric_limits<int>::max()), update_cost(graph->edges.size(),
            std::numeric_limits<int>::max());

    mask.at(source_node) = true;
    cost.at(source_node) = 0;
    update_cost.at(source_node) = 0;

    int *d_edges = nullptr;
    int *d_destinations = nullptr;
    int *d_weights = nullptr;
    int *d_previous_node = nullptr;
    int *d_mask = nullptr;
    int *d_cost = nullptr;
    int *d_update_cost = nullptr;

    M_C(hipMalloc((void**) &d_edges,          graph->edges.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_destinations,   graph->destinations.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_weights,        graph->weights.size() * sizeof(int)));

    M_C(hipMalloc((void**) &d_previous_node, previous_nodes.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_mask, mask.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_cost, cost.size() * sizeof(int)));
    M_C(hipMalloc((void**) &d_update_cost, update_cost.size() * sizeof(int)));

    M_C(hipMemcpy(d_edges,        &graph->edges[0],        graph->edges.size() * sizeof(int),          hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_destinations, &graph->destinations[0], graph->destinations.size() * sizeof(int),   hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_weights,      &graph->weights[0],      graph->weights.size() * sizeof(int),        hipMemcpyHostToDevice));

    M_C(hipMemcpy(d_previous_node,&previous_nodes[0],  previous_nodes.size() * sizeof(int),hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_mask,         &mask[0],            mask.size() * sizeof(int),          hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_cost,         &cost[0],            cost.size() * sizeof(int),          hipMemcpyHostToDevice));
    M_C(hipMemcpy(d_update_cost,  &update_cost[0],     update_cost.size() * sizeof(int),   hipMemcpyHostToDevice));

    // while we still find false in the mask (Ma not empty)
    while (std::find(mask.begin(), mask.end(), true) != mask.end())
    {
        int numBlocks = ceil((double)graph->edges.size() / 1024);

        dim3 threadsPerBlock(32, 32);
        M_CFUN((CUDA_SSSP_Kernel1<<<numBlocks, threadsPerBlock>>>(d_edges, d_destinations, d_weights,
                d_previous_node, d_mask, d_cost, d_update_cost, graph->edges.size(), graph->destinations.size())));

        M_CFUN((CUDA_SSSP_Kernel2<<<numBlocks, threadsPerBlock>>>(d_mask, d_cost, d_update_cost, graph->edges.size())));

        //copy back mask
        M_C(hipMemcpy(&mask[0], d_mask, mask.size() * sizeof(int), hipMemcpyDeviceToHost));
    }

    M_C(hipMemcpy(&previous_nodes[0], d_previous_node, previous_nodes.size() * sizeof(int), hipMemcpyDeviceToHost));
    M_C(hipMemcpy(&cost[0], d_cost, cost.size() * sizeof(int), hipMemcpyDeviceToHost));
    M_C(hipMemcpy(&update_cost[0], d_update_cost, update_cost.size() * sizeof(int), hipMemcpyDeviceToHost));

    M_C(hipFree(d_edges));
    M_C(hipFree(d_destinations));
    M_C(hipFree(d_weights));
    M_C(hipFree(d_previous_node));
    M_C(hipFree(d_mask));
    M_C(hipFree(d_cost));
    M_C(hipFree(d_update_cost));

    std::shared_ptr<Paths> paths = std::make_shared<Paths>(Paths(previous_nodes, source_node, graph));

    return paths;
}